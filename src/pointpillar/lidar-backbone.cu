/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_fp16.h>

#include <numeric>

#include "lidar-backbone.hpp"
#include "common/check.hpp"
#include "common/launch.cuh"
#include "common/tensorrt.hpp"
#include <fstream>
#include <iostream>

namespace pointpillar {
namespace lidar {

class BackboneImplement : public Backbone {
public:
    virtual ~BackboneImplement() {
        if (cls_) checkRuntime(hipFree(cls_));
        if (box_) checkRuntime(hipFree(box_));
        if (dir_) checkRuntime(hipFree(dir_));
    }

    bool init(const std::string& model) {
        engine_ = TensorRT::load(model);
        if (engine_ == nullptr) return false;

        cls_dims_ = engine_->static_dims(3);
        box_dims_ = engine_->static_dims(4);
        dir_dims_ = engine_->static_dims(5);

        int32_t volumn = std::accumulate(cls_dims_.begin(), cls_dims_.end(), 1, std::multiplies<int32_t>());
        checkRuntime(hipMalloc(&cls_, volumn * sizeof(float)));

        volumn = std::accumulate(box_dims_.begin(), box_dims_.end(), 1, std::multiplies<int32_t>());
        checkRuntime(hipMalloc(&box_, volumn * sizeof(float)));

        volumn = std::accumulate(dir_dims_.begin(), dir_dims_.end(), 1, std::multiplies<int32_t>());
        checkRuntime(hipMalloc(&dir_, volumn * sizeof(float)));
        return true;
    }

    virtual void print() override { engine_->print("Lidar Backbone"); }

    virtual void forward(const nvtype::half* voxels, const unsigned int* voxel_idxs, const unsigned int* params, void* stream = nullptr) override {
        hipStream_t _stream = reinterpret_cast<hipStream_t>(stream);
        engine_->forward({voxels, voxel_idxs, params, cls_, box_, dir_}, static_cast<hipStream_t>(_stream));

        // 以下代码是自己加的
        // 保存anchor
        // float *h_voxels_ = nullptr;
        // float *h_voxel_idxs = nullptr;
        // float *h_voxel_num = nullptr;
        // std::vector<int> voxel_idxs_dims,voxel_num_dims,voxels_dims;
        // voxels_dims=engine_->static_dims(0);
        // voxel_idxs_dims=engine_->static_dims(1);
        // voxel_num_dims=engine_->static_dims(2);

        // int32_t volumn0 = std::accumulate(voxels_dims.begin(), voxels_dims.end(), 1, std::multiplies<int32_t>());
        // int32_t volumn1 = std::accumulate(voxel_idxs_dims.begin(), voxel_idxs_dims.end(), 1, std::multiplies<int32_t>());
        // int32_t volumn2 = std::accumulate(voxel_num_dims.begin(), voxel_num_dims.end(), 1, std::multiplies<int32_t>());

        // // checkRuntime(hipHostMalloc(&h_voxels_, volumn0 * sizeof(float)));
        // // checkRuntime(hipMemcpy(h_voxels_, voxels, volumn0 * sizeof(float), hipMemcpyDeviceToHost));
        // // std::ofstream out("voxels.txt");
        // // if (out.is_open()) {
        // //     for (int i = 0; i < volumn0; i++) out << h_voxels_[i] << std::endl;
        // //     out.close();
        // // }

        // // int a=1;

        
        //volumn3=41997322=BHWC_cls=1x248x432x392
        int32_t volumn_cls = std::accumulate(cls_dims_.begin(), cls_dims_.end(), 1, std::multiplies<int32_t>());
        //volumn4=20998656=BHWC_box=1x248x432x196
        int32_t volumn4 = std::accumulate(box_dims_.begin(), box_dims_.end(), 1, std::multiplies<int32_t>());
        //volumn4=5999616=BHWC_dir=1x248x432x56
        int32_t volumn5 = std::accumulate(dir_dims_.begin(), dir_dims_.end(), 1, std::multiplies<int32_t>());
        
        float *h_cls = nullptr;
        // float *h_box = nullptr;;
        // float *h_dir = nullptr;;

        // int32_t volumn = std::accumulate(cls_dims_.begin(), cls_dims_.end(), 1, std::multiplies<int32_t>());
        checkRuntime(hipHostMalloc(&h_cls, volumn_cls * sizeof(float)));
        checkRuntime(hipMemcpy(h_cls, cls_, volumn_cls * sizeof(float), hipMemcpyDeviceToHost));

        std::ofstream out("../data_output/4d_array_trt.bin", std::ios::binary);
        if (out) {
            out.write(reinterpret_cast<const char*>(h_cls), volumn_cls*sizeof(h_cls));
            std::cout<<"!!!!!!!!!!!!!!!!!!!!!!"<<std::endl;
            std::cout << "Binary file saved to: data/4d_array_trt.bin" << std::endl;
            std::cout<<"!!!!!!!!!!!!!!!!!!!!!!"<<std::endl;
        } else {
            std::cerr << "Error: Failed to save file!" << std::endl;
            // return 1;
        }
        // std::ofstream out("cls.txt");
        // if (out.is_open()) {
        //     for (int i = 0; i < volumn; i++) out << h_cls[i] << std::endl;
        //     out.close();
        // }


        // volumn = std::accumulate(box_dims_.begin(), box_dims_.end(), 1, std::multiplies<int32_t>());
        // checkRuntime(hipHostMalloc(&h_box, volumn * sizeof(float)));
        // checkRuntime(hipMemcpy(h_box, box_, volumn * sizeof(float), hipMemcpyDeviceToHost));
        // std::ofstream out2("box.txt");
        // if (out2.is_open()) {
        //     for (int i = 0; i < volumn; i++) out2 << h_box[i] << std::endl;
        //     out2.close();
        // }

        // volumn = std::accumulate(dir_dims_.begin(), dir_dims_.end(), 1, std::multiplies<int32_t>());
        // checkRuntime(hipHostMalloc(&h_dir, volumn * sizeof(float)));
        // checkRuntime(hipMemcpy(h_dir, dir_, volumn * sizeof(float), hipMemcpyDeviceToHost));
        // std::ofstream out3("dir.txt");
        // if (out3.is_open()) {
        //     for (int i = 0; i < volumn; i++) out3 << h_dir[i] << std::endl;
        //     out3.close();
        // }

        //         // 释放内存
        // if (h_cls) hipHostFree(h_cls);
        // if (h_box) hipHostFree(h_box);
        // if (h_dir) hipHostFree(h_dir);

    }

    virtual float* cls() override { return cls_; }
    virtual float* box() override { return box_; }
    virtual float* dir() override { return dir_; }

private:
    std::shared_ptr<TensorRT::Engine> engine_;
    float *cls_ = nullptr;
    float *box_ = nullptr;
    float *dir_ = nullptr;
    std::vector<int> cls_dims_, box_dims_, dir_dims_;
};

std::shared_ptr<Backbone> create_backbone(const std::string& model) {
  std::shared_ptr<BackboneImplement> instance(new BackboneImplement());
  if (!instance->init(model)) {
    instance.reset();
  }
  return instance;
}

};  // namespace lidar
};  // namespace pointpillar